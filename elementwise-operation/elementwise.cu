#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAddKernel(float *a, float *b, float *c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        c[i] = a[i] + b[i];
    }
}

__global__ void vectorSubKernel(float *a, float *b, float *c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        c[i] = a[i] - b[i];
    }
}

__global__ void vectorMulKernel(float *a, float *b, float *c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        c[i] = a[i] * b[i];
    }
}

extern "C"{
    float *VectorAdd(float *arrA, float *arrB, int n){
        float *h_c;
        float *d_a, *d_b, *d_c;

        hipHostMalloc((void **) &h_c, sizeof(float) * n, hipHostMallocDefault);
        hipMalloc((void **) &d_c, sizeof(float) * n);
        hipMalloc((void **) &d_a, sizeof(float) * n);
        hipMalloc((void **) &d_b, sizeof(float) * n);

        hipMemcpy(d_a, arrA, sizeof(float) * n, hipMemcpyHostToDevice);
        hipMemcpy(d_b, arrB, sizeof(float) * n, hipMemcpyHostToDevice);

        vectorAddKernel <<< 2, (n+1/2) >>> (d_a, d_b, d_c, n);
        hipDeviceSynchronize();

        hipMemcpy(h_c, d_c, sizeof(float) * n, hipMemcpyDeviceToHost);
        hipFree(d_c);
        hipFree(d_a);
        hipFree(d_b);

        return h_c;
    }

    float *VectorSub(float *arrA, float *arrB, int n){
        float *h_c;
        float *d_a, *d_b, *d_c;

        hipHostMalloc((void **) &h_c, sizeof(float) * n, hipHostMallocDefault);
        hipMalloc((void **) &d_c, sizeof(float) * n);
        hipMalloc((void **) &d_a, sizeof(float) * n);
        hipMalloc((void **) &d_b, sizeof(float) * n);

        hipMemcpy(d_a, arrA, sizeof(float) * n, hipMemcpyHostToDevice);
        hipMemcpy(d_b, arrB, sizeof(float) * n, hipMemcpyHostToDevice);

        vectorSubKernel <<< 2, (n+1/2) >>> (d_a, d_b, d_c, n);
        hipDeviceSynchronize();

        hipMemcpy(h_c, d_c, sizeof(float) * n, hipMemcpyDeviceToHost);
        hipFree(d_c);
        hipFree(d_a);
        hipFree(d_b);

        return h_c;
    }

    float *VectorMul(float *arrA, float *arrB, int n){
        float *h_c;
        float *d_a, *d_b, *d_c;

        hipHostMalloc((void **) &h_c, sizeof(float) * n, hipHostMallocDefault);
        hipMalloc((void **) &d_c, sizeof(float) * n);
        hipMalloc((void **) &d_a, sizeof(float) * n);
        hipMalloc((void **) &d_b, sizeof(float) * n);

        hipMemcpy(d_a, arrA, sizeof(float) * n, hipMemcpyHostToDevice);
        hipMemcpy(d_b, arrB, sizeof(float) * n, hipMemcpyHostToDevice);

        vectorMulKernel <<< 2, (n+1/2) >>> (d_a, d_b, d_c, n);
        hipDeviceSynchronize();

        hipMemcpy(h_c, d_c, sizeof(float) * n, hipMemcpyDeviceToHost);
        hipFree(d_c);
        hipFree(d_a);
        hipFree(d_b);

        return h_c;
    }
}