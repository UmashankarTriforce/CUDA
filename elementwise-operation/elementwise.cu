#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAddKernel(double *a, double *b, double *c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        c[i] = a[i] + b[i];
    }
}

__global__ void vectorSubKernel(double *a, double *b, double *c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        c[i] = a[i] - b[i];
    }
}

__global__ void vectorMulKernel(double *a, double *b, double *c, int n){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        c[i] = a[i] * b[i];
    }
}

extern "C"{
    double *VectorAdd(double *arrA, double *arrB, int n){
        double *h_c;
        double *d_a, *d_b, *d_c;

        hipHostMalloc((void **) &h_c, sizeof(double) * n, hipHostMallocDefault);
        hipMalloc((void **) &d_c, sizeof(double) * n);
        hipMalloc((void **) &d_a, sizeof(double) * n);
        hipMalloc((void **) &d_b, sizeof(double) * n);

        hipMemcpy(d_a, arrA, sizeof(double) * n, hipMemcpyHostToDevice);
        hipMemcpy(d_b, arrB, sizeof(double) * n, hipMemcpyHostToDevice);

        vectorAddKernel <<< 2, (n+1/2) >>> (d_a, d_b, d_c, n);
        hipDeviceSynchronize();

        hipMemcpy(h_c, d_c, sizeof(double) * n, hipMemcpyDeviceToHost);
        hipFree(d_c);
        hipFree(d_a);
        hipFree(d_b);

        return h_c;
    }

    double *VectorSub(double *arrA, double *arrB, int n){
        double *h_c;
        double *d_a, *d_b, *d_c;

        hipHostMalloc((void **) &h_c, sizeof(double) * n, hipHostMallocDefault);
        hipMalloc((void **) &d_c, sizeof(double) * n);
        hipMalloc((void **) &d_a, sizeof(double) * n);
        hipMalloc((void **) &d_b, sizeof(double) * n);

        hipMemcpy(d_a, arrA, sizeof(double) * n, hipMemcpyHostToDevice);
        hipMemcpy(d_b, arrB, sizeof(double) * n, hipMemcpyHostToDevice);

        vectorSubKernel <<< 2, (n+1/2) >>> (d_a, d_b, d_c, n);
        hipDeviceSynchronize();

        hipMemcpy(h_c, d_c, sizeof(double) * n, hipMemcpyDeviceToHost);
        hipFree(d_c);
        hipFree(d_a);
        hipFree(d_b);

        return h_c;
    }

    double *VectorMul(double *arrA, double *arrB, int n){
        double *h_c;
        double *d_a, *d_b, *d_c;

        hipHostMalloc((void **) &h_c, sizeof(double) * n, hipHostMallocDefault);
        hipMalloc((void **) &d_c, sizeof(double) * n);
        hipMalloc((void **) &d_a, sizeof(double) * n);
        hipMalloc((void **) &d_b, sizeof(double) * n);

        hipMemcpy(d_a, arrA, sizeof(double) * n, hipMemcpyHostToDevice);
        hipMemcpy(d_b, arrB, sizeof(double) * n, hipMemcpyHostToDevice);

        vectorMulKernel <<< 2, (n+1/2) >>> (d_a, d_b, d_c, n);
        hipDeviceSynchronize();

        hipMemcpy(h_c, d_c, sizeof(double) * n, hipMemcpyDeviceToHost);
        hipFree(d_c);
        hipFree(d_a);
        hipFree(d_b);

        return h_c;
    }
}