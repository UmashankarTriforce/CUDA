
#include <hip/hip_runtime.h>
#include <hipblas.h>

extern "C"{
    double *DGEMM(double *A, double *B, int rowA, int colA, int rowB, int colB){
        double *devA, *devB, *devC, *hostC;

        hipMalloc((void **) &devA, sizeof(double) * (rowA * colA));
        hipMalloc((void **) &devB, sizeof(double) * (rowB * colB));
        hipMalloc((void **) &devC, sizeof(double) * (rowA * colB));
        hipHostMalloc((void **) &hostC, sizeof(double) * (rowA * colB), hipHostMallocDefault);
        hipMemcpy(devA, A, sizeof(double) * (rowA * colA), hipMemcpyHostToDevice);
        hipMemcpy(devB, B, sizeof(double) * (rowB * colB), hipMemcpyHostToDevice);

        hipblasHandle_t handle;
        hipblasStatus_t status = hipblasCreate(&handle);

        double alpha = 1.0f;double beta = 0.0f;
        status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, rowA, colB,\
            colA, &alpha, devA, colA, devB, colB, &beta,  devC, rowA);
        status = hipblasDestroy(handle);
        hipMemcpy(hostC, devC, sizeof(double) * (rowA * colB), hipMemcpyDeviceToHost);

        hipFree(devA);
        hipFree(devB);
        hipFree(devC);

        return hostC;
    }
}